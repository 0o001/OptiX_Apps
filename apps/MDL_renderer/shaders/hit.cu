#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"
#include "per_ray_data.h"
#include "vertex_attributes.h"
#include "function_indices.h"
#include "material_definition_mdl.h"
#include "light_definition.h"
#include "shader_common.h"
#include "transform.h"
#include "random_number_generators.h"

// Contained in per_ray_data.h:
//#include <mi/neuraylib/target_code_types.h>

// The MDL texture runtime functions: texture, MBSDF, light profile, and scene data (dummy) lookup functions.
// These are declared extern and can only appear in one module inside the pipeline or there will be OptiX compilation errors.
// Means all functions potentially accessing any of these MDL runtime functions must be implemented in this module.
// That's the reason why the arbitrary mesh light sampling routine is here and not in light_sample.cu
#define TEX_SUPPORT_NO_VTABLES
#define TEX_SUPPORT_NO_DUMMY_SCENEDATA
#include "texture_lookup.h"

// This renderer is not implementing support for derivatives (ray differentials).
// It only needs this Shading_state_materialy structure without derivatives support.
typedef mi::neuraylib::Shading_state_material Mdl_state;


// DEBUG Helper code.
//uint3 theLaunchIndex = optixGetLaunchIndex();
//if (theLaunchIndex.x == 256 && theLaunchIndex.y == 256)
//{
//  printf("value = %f\n", value);
//}

//thePrd->radiance += make_float3(value);
//thePrd->eventType = mi::neuraylib::BSDF_EVENT_ABSORB;
//return;


extern "C" __constant__ SystemData sysData;

// This shader handles every supported feature of the the renderer.
extern "C" __global__ void __closesthit__radiance()
{
  GeometryInstanceData theData = sysData.geometryInstanceData[optixGetInstanceId()];
  // theData.ids: .x = idMaterial, .y = idLight, .z = idObject

  const unsigned int thePrimitiveIndex = optixGetPrimitiveIndex();

  // Cast the hipDeviceptr_t to the actual format of the Triangles attributes and indices.
  const uint3* indices = reinterpret_cast<uint3*>(theData.indices);
  const uint3  tri     = indices[thePrimitiveIndex];

  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(theData.attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  const float2 theBarycentrics = optixGetTriangleBarycentrics(); // beta and gamma
  const float  alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;

  float4 objectToWorld[3];
  float4 worldToObject[3];

  getTransforms(optixGetTransformListHandle(0), objectToWorld, worldToObject); // Single instance level transformation list only.

  float3 ng = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex);
  float3 tg = attr0.tangent  * alpha + attr1.tangent  * theBarycentrics.x + attr2.tangent  * theBarycentrics.y;
  float3 ns = attr0.normal   * alpha + attr1.normal   * theBarycentrics.x + attr2.normal   * theBarycentrics.y;
  
  const float3 tc = attr0.texcoord * alpha + attr1.texcoord * theBarycentrics.x + attr2.texcoord * theBarycentrics.y;

  // Transform into internal space == world space.
  ng = normalize(transformNormal(worldToObject, ng));
  tg = normalize(transformVector(objectToWorld, tg));
  ns = normalize(transformNormal(worldToObject, ns));

  TBN tbn(tg, ns); // Calculate an otho-normal system respective to the shading normal.

  // Get the current rtPayload pointer from the unsigned int payload registers p0 and p1.
  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  thePrd->flags |= FLAG_HIT; // Required to distinguish surface hits from random walk miss.

  thePrd->distance = optixGetRayTmax(); // Return the current path segment distance, needed for absorption calculations in the integrator.
  
  // PRECISION Calculate this from the object space vertex positions and transform to world for better accuracy when needed.
  // Same as: thePrd->pos = optixGetWorldRayOrigin() + optixGetWorldRayDirection() * optixGetRayTmax();
  thePrd->pos += thePrd->wi * thePrd->distance; 

  // If we're inside a volume and hit something, the path throughput needs to be modulated
  // with the transmittance along this segment before adding surface or light radiance!
  if (0 < thePrd->idxStack) // This assumes the first stack entry is vaccuum.
  {
    thePrd->throughput *= expf(thePrd->sigma_t * -thePrd->distance);

    // Increment the volume scattering random walk counter.
    // Unused when FLAG_VOLUME_SCATTERING is not set.
    ++thePrd->walk;
  }

  Mdl_state state;

  // The result of state::normal(). It represents the shading normal as determined by the renderer.
  // This field will be updated to the result of "geometry.normal" by the material or BSDF init functions,
  // if requested during code generation with set_option("include_geometry_normal", true) which is the default.
  state.normal = tbn.normal;

  // The result of state::geometry_normal().
  // It represents the geometry normal as determined by the renderer.
  state.geom_normal = ng;

  // The result of state::position().
  // It represents the position where the material should be evaluated.
  state.position = thePrd->pos;

  // The result of state::animation_time().
  // It represents the time of the current sample in seconds.
  state.animation_time = 0.0f; // This renderer implements no support for animations.

  // An array containing the results of state::texture_coordinate(i).
  // The i-th entry represents the texture coordinates of the i-th texture space at the current position.
  // Only one element here because "num_texture_spaces" option has been set to 1.
  state.text_coords = &tc;

  // An array containing the results of state::texture_tangent_u(i).
  // The i-th entry represents the texture tangent vector of the i-th texture space at the
  // current position, which points in the direction of the projection of the tangent to the
  // positive u axis of this texture space onto the plane defined by the original surface normal.
  // Only one element because "num_texture_spaces" option has been set to 1.
  state.tangent_u = &tbn.tangent;

  // An array containing the results of state::texture_tangent_v(i).
  // The i-th entry represents the texture bitangent vector of the i-th texture space at the
  // current position, which points in the general direction of the positive v axis of this
  // texture space, but is orthogonal to both the original surface normal and the tangent
  // of this texture space.
  // Only one element because "num_texture_spaces" option has been set to 1.
  state.tangent_v = &tbn.bitangent;

  // The texture results lookup table.
  // The size must match the backend set_option("num_texture_results") value.
  // Values will be modified by the init functions to avoid duplicate texture fetches 
  // and duplicate calculation of values (texture coordinate system).
  // This implementation is using the single material init function, not the individual init per distribution function.
  // PERF This influences how many things can be precalculated inside the init() function.
  // If the number of result elements in this array is lower than what is required,
  // the expressions for the remaining results will be compiled into the sample() and eval() functions
  // which will make the compilation and runtime performance slower. 
  // For very resource-heavy materials, experiment with bigger arrays.
  float4 texture_results[16];

  state.text_results = texture_results;

  // A pointer to a read-only data segment.
  // For "PTX", "LLVM-IR" and "native" JIT backend.
  // For other backends, this should be NULL.
  state.ro_data_segment = nullptr;

  // A 4x4 transformation matrix in row-major order transforming from world to object coordinates.
  // The last row is always implied to be (0, 0, 0, 1) and does not have to be provided.
  // It is used by the state::transform_*() methods.
  // This field is only used if the uniform state is included.
  state.world_to_object = worldToObject;

  // A 4x4 transformation matrix in row-major order transforming from object to world coordinates.
  // The last row is always implied to be (0, 0, 0, 1) and does not have to be provided.
  // It is used by the state::transform_*() methods.
  // This field is only used if the uniform state is included.
  state.object_to_world = objectToWorld;

  // The result of state::object_id().
  // It is an application-specific identifier of the hit object as provided in a scene.
  // It can be used to make instanced objects look different in spite of the same used material.
  // This field is only used if the uniform state is included.
  state.object_id = theData.ids.z; // idObject, this is the sg::Instance node ID.

  // The result of state::meters_per_scene_unit().
  // The field is only used if the "fold_meters_per_scene_unit" option is set to false.
  // Otherwise, the value of the "meters_per_scene_unit" option will be used in the code.
  state.meters_per_scene_unit = 1.0f;
    
  const MaterialDefinitionMDL& material = sysData.materialDefinitionsMDL[theData.ids.x];

  mi::neuraylib::Resource_data res_data = { nullptr, material.texture_handler };

  const DeviceShaderConfiguration& shaderConfiguration = sysData.shaderConfigurations[material.indexShader];

  // Using a single material init function instead of per distribution init functions.
  // This is always present, even if it just returns.
  optixDirectCall<void>(shaderConfiguration.idxCallInit, &state, &res_data, nullptr, material.arg_block);

  // Explicitly include edge-on cases as frontface condition!
  // Keeps the material stack from overflowing at silhouettes.
  // Prevents that silhouettes of thin-walled materials use the backface material.
  // Using the true geometry normal attribute as originally defined on the frontface!
  const bool isFrontFace = (0.0f <= dot(thePrd->wo, state.geom_normal));

  // thin_walled value in case the expression is a constant.
  bool thin_walled = ((shaderConfiguration.flags & IS_THIN_WALLED) != 0);

  if (0 <= shaderConfiguration.idxCallThinWalled)
  {
    optixDirectCall<void>(shaderConfiguration.idxCallThinWalled, &thin_walled, &state, &res_data, nullptr, material.arg_block);
  }

  // IOR value in case the material ior expression is constant.
  float3 ior = shaderConfiguration.ior;

  if (0 <= shaderConfiguration.idxCallIor)
  {
    optixDirectCall<void>(shaderConfiguration.idxCallIor, &ior, &state, &res_data, nullptr, material.arg_block);
  }

  // Handle optional surface and backface emission expressions.
  // Default to no EDF.
  int idxCallEmissionEval          = -1; 
  int idxCallEmissionIntensity     = -1;
  int idxCallEmissionIntensityMode = -1;
  // These are not used when there is no emission, no need to initialize.
  float3 emission_intensity;
  int    emission_intensity_mode;
  
  // MDL Specs: There is no emission on the back-side unless an EDF is specified with the backface field and thin_walled is set to true.
  if (isFrontFace)
  {
    idxCallEmissionEval          = shaderConfiguration.idxCallSurfaceEmissionEval;
    idxCallEmissionIntensity     = shaderConfiguration.idxCallSurfaceEmissionIntensity;
    idxCallEmissionIntensityMode = shaderConfiguration.idxCallSurfaceEmissionIntensityMode;
    
    emission_intensity      = shaderConfiguration.surface_intensity;
    emission_intensity_mode = shaderConfiguration.surface_intensity_mode;
  }
  else if (thin_walled) // && !isFrontFace
  {
    // These can be the same callable indices if the expressions from surface and backface were identical.
    idxCallEmissionEval          = shaderConfiguration.idxCallBackfaceEmissionEval;
    idxCallEmissionIntensity     = shaderConfiguration.idxCallBackfaceEmissionIntensity;
    idxCallEmissionIntensityMode = shaderConfiguration.idxCallBackfaceEmissionIntensityMode;

    emission_intensity      = shaderConfiguration.backface_intensity;
    emission_intensity_mode = shaderConfiguration.backface_intensity_mode;
  }

  // Check if the hit geometry contains any emission.
  if (0 <= idxCallEmissionEval) 
  {
    if (0 <= idxCallEmissionIntensity) // Emission intensity is not a constant.
    {
      optixDirectCall<void>(idxCallEmissionIntensity, &emission_intensity, &state, &res_data, nullptr, material.arg_block);
    }
    if (0 <= idxCallEmissionIntensityMode) // Emission intensity mode is not a constant.
    {
      optixDirectCall<void>(idxCallEmissionIntensityMode, &emission_intensity_mode, &state, &res_data, nullptr, material.arg_block);
    }
    if (isNotNull(emission_intensity))
    {
      mi::neuraylib::Edf_evaluate_data<mi::neuraylib::DF_HSM_NONE> eval_data;

      eval_data.k1 = thePrd->wo; // input: outgoing direction (-ray.direction)
      //eval_data.cos : output: dot(normal, k1)
      //eval_data.edf : output: edf
      //eval_data.pdf : output: pdf (non-projected hemisphere)

      optixDirectCall<void>(idxCallEmissionEval, &eval_data, &state, &res_data, nullptr, material.arg_block);

      const float area = sysData.lightDefinitions[theData.ids.y].area; // This must be a mesh light, and then it has a valid idLight.

      eval_data.pdf = thePrd->distance * thePrd->distance / (area * eval_data.cos); // Solid angle measure.

      float weightMIS = 1.0f;
      // If the last event was diffuse or glossy, calculate the opposite MIS weight for this implicit light hit.
      if (sysData.directLighting && (thePrd->eventType & (mi::neuraylib::BSDF_EVENT_DIFFUSE | mi::neuraylib::BSDF_EVENT_GLOSSY)))
      {
        weightMIS = balanceHeuristic(thePrd->pdf, eval_data.pdf);
      }

      // Power (flux) [W] divided by light area gives radiant exitance [W/m^2].
      const float factor = (emission_intensity_mode == 0) ? 1.0f : 1.0f / area;
      
      thePrd->radiance += thePrd->throughput * emission_intensity * eval_data.edf * (factor * weightMIS);
    }
  }

  // Start fresh with the next BSDF sample.
  // Save the current path throughput for the direct lighting contribution.
  // The path throughput will be modulated with the BSDF sampling results before that.
  const float3 throughput = thePrd->throughput;
  // The pdf of the previous event was needed for the emission calculation above.
  thePrd->pdf = 0.0f;

  // Determine which BSDF to use when the material is thin-walled. 
  int idxCallScatteringSample = shaderConfiguration.idxCallSurfaceScatteringSample;
  int idxCallScatteringEval   = shaderConfiguration.idxCallSurfaceScatteringEval;

  // thin-walled and looking at the backface and backface.scattering expression available?
  if (thin_walled && !isFrontFace && 0 <= shaderConfiguration.idxCallBackfaceScatteringSample)
  {
    // Use the backface.scattering BSDF sample and evaluation functions.
    // Apparently the MDL code can handle front- and backfacing calculations appropriately with the original state and the properly setup volume IORs.
    // No need to flip normals to the ray side.
    idxCallScatteringSample = shaderConfiguration.idxCallBackfaceScatteringSample;
    idxCallScatteringEval   = shaderConfiguration.idxCallBackfaceScatteringEval; // Assumes both are valid.
  }

  // Importance sample the BSDF. 
  if (0 <= idxCallScatteringSample)
  {
    mi::neuraylib::Bsdf_sample_data sample_data;

    int idx = thePrd->idxStack;

    // If the hit is either on the surface or a thin-walled material,
    // the ray is inside the surrounding material and the material ior is on the other side.
    if (isFrontFace || thin_walled)
    {
      sample_data.ior1 = thePrd->stack[idx].ior; // From surrounding medium ior
      sample_data.ior2 = ior;                    // to material ior.
    }
    else 
    {
      // When hitting the backface of a non-thin-walled material, 
      // the ray is inside the current material and the surrounding material is on the other side.
      // The material's IOR is the current top-of-stack. We need the one further down!
      idx = max(0, idx - 1);

      sample_data.ior1 = ior;                    // From material ior 
      sample_data.ior2 = thePrd->stack[idx].ior; // to surrounding medium ior
    }
    sample_data.k1 = thePrd->wo; // == -optixGetWorldRayDirection()
    sample_data.xi = rng4(thePrd->seed);

    optixDirectCall<void>(idxCallScatteringSample, &sample_data, &state, &res_data, nullptr, material.arg_block);

    thePrd->wi          = sample_data.k2;            // Continuation direction.
    thePrd->throughput *= sample_data.bsdf_over_pdf; // Adjust the path throughput for all following incident lighting.
    thePrd->pdf         = sample_data.pdf;           // Note that specular events return pdf == 0.0f! (=> Not a path termination condition.)
    thePrd->eventType   = sample_data.event_type;    // This replaces the PRD flags used inside the other examples.
  }
  else
  {
    // If there is no valid scattering BSDF, it's the black bsdf() which ends the path.
    // This is usually happening with arbitrary mesh lights when only specifying emission.
    thePrd->eventType = mi::neuraylib::BSDF_EVENT_ABSORB;
    // None of the following code will have any effect in that case.
    return;
  }

  // Direct lighting if the sampled BSDF was diffuse and any light is in the scene.
  const int numLights = sysData.numLights;

  if (sysData.directLighting && 0 < numLights && (thePrd->eventType & (mi::neuraylib::BSDF_EVENT_DIFFUSE | mi::neuraylib::BSDF_EVENT_GLOSSY)))
  {
    // Sample one of many lights.
    // The caller picks the light to sample. Make sure the index stays in the bounds of the sysData.lightDefinitions array.
    const int indexLight = (1 < numLights) ? clamp(static_cast<int>(floorf(rng(thePrd->seed) * numLights)), 0, numLights - 1) : 0;
    
    const LightDefinition& light = sysData.lightDefinitions[indexLight];
    
    LightSample lightSample = optixDirectCall<LightSample, const LightDefinition&, PerRayData*>(NUM_LENS_TYPES + light.typeLight, light, thePrd);

    if (0.0f < lightSample.pdf && 0 <= idxCallScatteringEval)
    {
      mi::neuraylib::Bsdf_evaluate_data<mi::neuraylib::DF_HSM_NONE> eval_data;

      int idx = thePrd->idxStack;
      
      if (isFrontFace || thin_walled)
      {
        eval_data.ior1 = thePrd->stack[idx].ior;
        eval_data.ior2 = ior;
      }
      else
      {
        idx = max(0, idx - 1);

        eval_data.ior1 = ior;
        eval_data.ior2 = thePrd->stack[idx].ior;
      }
      
      eval_data.k1 = thePrd->wo;
      eval_data.k2 = lightSample.direction;

      optixDirectCall<void>(idxCallScatteringEval, &eval_data, &state, &res_data, nullptr, material.arg_block);

      // This already contains the fabsf(dot(lightSample.direction, state.normal)) factor!
      // For a white Lambert material, the bxdf components match the eval_data.pdf
      const float3 bxdf = eval_data.bsdf_diffuse + eval_data.bsdf_glossy;

      if (0.0f < eval_data.pdf && isNotNull(bxdf))
      {
        // Pass the current payload registers through to the shadow ray.
        unsigned int p0 = optixGetPayload_0();
        unsigned int p1 = optixGetPayload_1();

        thePrd->flags &= ~FLAG_SHADOW; // Clear the shadow flag.

        // Note that the sysData.sceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval 
        // to prevent self-intersections with the actual light geometry in the scene.
        optixTrace(sysData.topObject,
                   thePrd->pos, lightSample.direction, // origin, direction
                   sysData.sceneEpsilon, lightSample.distance - sysData.sceneEpsilon, 0.0f, // tmin, tmax, time
                   OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT, // The shadow ray type only uses anyhit programs.
                   TYPE_RAY_SHADOW, NUM_RAY_TYPES, TYPE_RAY_SHADOW,
                   p0, p1); // Pass through thePrd to the shadow ray.

        if ((thePrd->flags & FLAG_SHADOW) == 0) // Shadow flag not set?
        {
          const float weightMIS = (TYPE_LIGHT_POINT <= light.typeLight) ? 1.0f : balanceHeuristic(lightSample.pdf, eval_data.pdf);
          
          // The sampled emission needs to be scaled by the inverse probability to have selected this light,
          // Selecting one of many lights means the inverse of 1.0f / numLights.
          // This is using the path throughput before the sampling modulated it above.
          thePrd->radiance += throughput * bxdf * lightSample.radiance_over_pdf * (float(numLights) * weightMIS);
        }
      } 
    }
  }

  // Now after everything has been handled using the current material stack,
  // adjust the material stack if there was a transmission crossing a boundary surface.
  if (!thin_walled && (thePrd->eventType & mi::neuraylib::BSDF_EVENT_TRANSMISSION) != 0)
  {
    if (isFrontFace) // Entered a volume. 
    {
      float3 absorption = shaderConfiguration.absorption_coefficient;
      if (0 < shaderConfiguration.idxCallVolumeAbsorptionCoefficient)
      {
        optixDirectCall<void>(shaderConfiguration.idxCallVolumeAbsorptionCoefficient, &absorption, &state, &res_data, nullptr, material.arg_block);
      }

      float3 scattering = shaderConfiguration.scattering_coefficient;
      if (0 < shaderConfiguration.idxCallVolumeScatteringCoefficient)
      {
        optixDirectCall<void>(shaderConfiguration.idxCallVolumeScatteringCoefficient, &scattering, &state, &res_data, nullptr, material.arg_block);
      }

      float bias = shaderConfiguration.directional_bias;
      if (0 < shaderConfiguration.idxCallVolumeDirectionalBias)
      {
        optixDirectCall<void>(shaderConfiguration.idxCallVolumeDirectionalBias, &bias, &state, &res_data, nullptr, material.arg_block);
      }

      const int idx = min(thePrd->idxStack + 1, MATERIAL_STACK_LAST); // Push current medium parameters.

      thePrd->idxStack = idx;
      thePrd->stack[idx].ior     = ior;
      thePrd->stack[idx].sigma_a = absorption;
      thePrd->stack[idx].sigma_s = scattering;
      thePrd->stack[idx].bias    = bias;
      
      thePrd->sigma_t = absorption + scattering; // Update the current extinction coefficient.
    }
    else // if !isFrontFace. Left a volume.
    {
      const int idx = max(0, thePrd->idxStack - 1); // Pop current medium parameters.

      thePrd->idxStack = idx; 

      thePrd->sigma_t = thePrd->stack[idx].sigma_a + thePrd->stack[idx].sigma_s; // Update the current extinction coefficient.
    }
   
    thePrd->walk = 0; // Reset the number of random walk steps taken when crossing any volume boundary.
  }
}


// PERF Identical to radiance shader above, but used for materials without emission, which is the majority of materials.
extern "C" __global__ void __closesthit__radiance_no_emission()
{
  GeometryInstanceData theData = sysData.geometryInstanceData[optixGetInstanceId()];
  // theData.ids: .x = idMaterial, .y = idLight, .z = idObject

  const unsigned int thePrimitiveIndex = optixGetPrimitiveIndex();

  // Cast the hipDeviceptr_t to the actual format of the Triangles attributes and indices.
  const uint3* indices = reinterpret_cast<uint3*>(theData.indices);
  const uint3  tri     = indices[thePrimitiveIndex];

  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(theData.attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  const float2 theBarycentrics = optixGetTriangleBarycentrics(); // beta and gamma
  const float  alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;

  float4 objectToWorld[3];
  float4 worldToObject[3];

  getTransforms(optixGetTransformListHandle(0), objectToWorld, worldToObject); // Single instance level transformation list only.

  float3 ng = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex);
  float3 tg = attr0.tangent  * alpha + attr1.tangent  * theBarycentrics.x + attr2.tangent  * theBarycentrics.y;
  float3 ns = attr0.normal   * alpha + attr1.normal   * theBarycentrics.x + attr2.normal   * theBarycentrics.y;
  
  const float3 tc = attr0.texcoord * alpha + attr1.texcoord * theBarycentrics.x + attr2.texcoord * theBarycentrics.y;

  // Transform into internal space == world space.
  ng = normalize(transformNormal(worldToObject, ng));
  tg = normalize(transformVector(objectToWorld, tg));
  ns = normalize(transformNormal(worldToObject, ns));

  TBN tbn(tg, ns); // Calculate an otho-normal system respective to the shading normal.

  // Get the current rtPayload pointer from the unsigned int payload registers p0 and p1.
  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  thePrd->flags |= FLAG_HIT; // Required to distinguish surface hits from random walk miss.

  thePrd->distance = optixGetRayTmax(); // Return the current path segment distance, needed for absorption calculations in the integrator.
  
  // PRECISION Calculate this from the object space vertex positions and transform to world for better accuracy when needed.
  // Same as: thePrd->pos = optixGetWorldRayOrigin() + optixGetWorldRayDirection() * optixGetRayTmax();
  thePrd->pos += thePrd->wi * thePrd->distance; 

  // If we're inside a volume and hit something, the path throughput needs to be modulated
  // with the transmittance along this segment before adding surface or light radiance!
  if (0 < thePrd->idxStack) // This assumes the first stack entry is vaccuum.
  {
    thePrd->throughput *= expf(thePrd->sigma_t * -thePrd->distance);

    // Increment the volume scattering random walk counter.
    // Unused when FLAG_VOLUME_SCATTERING is not set.
    ++thePrd->walk;
  }

  Mdl_state state;

  float4 texture_results[16];

  // For explanations of these fields see comments inside __closesthit__radiance above.
  state.normal                = tbn.normal;
  state.geom_normal           = ng;
  state.position              = thePrd->pos;
  state.animation_time        = 0.0f;
  state.text_coords           = &tc;
  state.tangent_u             = &tbn.tangent;
  state.tangent_v             = &tbn.bitangent;
  state.text_results          = texture_results;
  state.ro_data_segment       = nullptr;
  state.world_to_object       = worldToObject;
  state.object_to_world       = objectToWorld;
  state.object_id             = theData.ids.z;
  state.meters_per_scene_unit = 1.0f;
    
  const MaterialDefinitionMDL& material = sysData.materialDefinitionsMDL[theData.ids.x];

  mi::neuraylib::Resource_data res_data = { nullptr, material.texture_handler };

  const DeviceShaderConfiguration& shaderConfiguration = sysData.shaderConfigurations[material.indexShader];

  // Using a single material init function instead of per distribution init functions.
  // This is always present, even if it just returns.
  optixDirectCall<void>(shaderConfiguration.idxCallInit, &state, &res_data, nullptr, material.arg_block);

  // Explicitly include edge-on cases as frontface condition!
  // Keeps the material stack from overflowing at silhouettes.
  // Prevents that silhouettes of thin-walled materials use the backface material.
  // Using the true geometry normal attribute as originally defined on the frontface!
  const bool isFrontFace = (0.0f <= dot(thePrd->wo, state.geom_normal));

  // thin_walled value in case the expression is a constant.
  bool thin_walled = ((shaderConfiguration.flags & IS_THIN_WALLED) != 0);

  if (0 <= shaderConfiguration.idxCallThinWalled)
  {
    optixDirectCall<void>(shaderConfiguration.idxCallThinWalled, &thin_walled, &state, &res_data, nullptr, material.arg_block);
  }

  // IOR value in case the material ior expression is constant.
  float3 ior = shaderConfiguration.ior;

  if (0 <= shaderConfiguration.idxCallIor)
  {
    optixDirectCall<void>(shaderConfiguration.idxCallIor, &ior, &state, &res_data, nullptr, material.arg_block);
  }

  // Start fresh with the next BSDF sample.
  // Save the current path throughput for the direct lighting contribution.
  // The path throughput will be modulated with the BSDF sampling results before that.
  const float3 throughput = thePrd->throughput;
  // The pdf of the previous event was needed for the emission calculation above.
  thePrd->pdf = 0.0f;

  // Determine which BSDF to use when the material is thin-walled. 
  int idxCallScatteringSample = shaderConfiguration.idxCallSurfaceScatteringSample;
  int idxCallScatteringEval   = shaderConfiguration.idxCallSurfaceScatteringEval;

  // thin-walled and looking at the backface and backface.scattering expression available?
  if (thin_walled && !isFrontFace && 0 <= shaderConfiguration.idxCallBackfaceScatteringSample)
  {
    // Use the backface.scattering BSDF sample and evaluation functions.
    // Apparently the MDL code can handle front- and backfacing calculations appropriately with the original state and the properly setup volume IORs.
    // No need to flip normals to the ray side.
    idxCallScatteringSample = shaderConfiguration.idxCallBackfaceScatteringSample;
    idxCallScatteringEval   = shaderConfiguration.idxCallBackfaceScatteringEval; // Assumes both are valid.
  }

  // Importance sample the BSDF. 
  if (0 <= idxCallScatteringSample)
  {
    mi::neuraylib::Bsdf_sample_data sample_data;

    int idx = thePrd->idxStack;

    // If the hit is either on the surface or a thin-walled material,
    // the ray is inside the surrounding material and the material ior is on the other side.
    if (isFrontFace || thin_walled)
    {
      sample_data.ior1 = thePrd->stack[idx].ior; // From surrounding medium ior
      sample_data.ior2 = ior;                    // to material ior.
    }
    else 
    {
      // When hitting the backface of a non-thin-walled material, 
      // the ray is inside the current material and the surrounding material is on the other side.
      // The material's IOR is the current top-of-stack. We need the one further down!
      idx = max(0, idx - 1);

      sample_data.ior1 = ior;                    // From material ior 
      sample_data.ior2 = thePrd->stack[idx].ior; // to surrounding medium ior
    }
    sample_data.k1 = thePrd->wo; // == -optixGetWorldRayDirection()
    sample_data.xi = rng4(thePrd->seed);

    optixDirectCall<void>(idxCallScatteringSample, &sample_data, &state, &res_data, nullptr, material.arg_block);

    thePrd->wi          = sample_data.k2;            // Continuation direction.
    thePrd->throughput *= sample_data.bsdf_over_pdf; // Adjust the path throughput for all following incident lighting.
    thePrd->pdf         = sample_data.pdf;           // Note that specular events return pdf == 0.0f! (=> Not a path termination condition.)
    thePrd->eventType   = sample_data.event_type;    // This replaces the PRD flags used inside the other examples.
  }
  else
  {
    // If there is no valid scattering BSDF, it's the black bsdf() which ends the path.
    // This is usually happening with arbitrary mesh lights when only specifying emission.
    thePrd->eventType = mi::neuraylib::BSDF_EVENT_ABSORB;
    // None of the following code will have any effect in that case.
    return;
  }

  // Direct lighting if the sampled BSDF was diffuse and any light is in the scene.
  const int numLights = sysData.numLights;

  if (sysData.directLighting && 0 < numLights && (thePrd->eventType & (mi::neuraylib::BSDF_EVENT_DIFFUSE | mi::neuraylib::BSDF_EVENT_GLOSSY)))
  {
    // Sample one of many lights.
    // The caller picks the light to sample. Make sure the index stays in the bounds of the sysData.lightDefinitions array.
    const int indexLight = (1 < numLights) ? clamp(static_cast<int>(floorf(rng(thePrd->seed) * numLights)), 0, numLights - 1) : 0;
    
    const LightDefinition& light = sysData.lightDefinitions[indexLight];
    
    LightSample lightSample = optixDirectCall<LightSample, const LightDefinition&, PerRayData*>(NUM_LENS_TYPES + light.typeLight, light, thePrd);

    if (0.0f < lightSample.pdf && 0 <= idxCallScatteringEval)
    {
      mi::neuraylib::Bsdf_evaluate_data<mi::neuraylib::DF_HSM_NONE> eval_data;

      int idx = thePrd->idxStack;
      
      if (isFrontFace || thin_walled)
      {
        eval_data.ior1 = thePrd->stack[idx].ior;
        eval_data.ior2 = ior;
      }
      else
      {
        idx = max(0, idx - 1);

        eval_data.ior1 = ior;
        eval_data.ior2 = thePrd->stack[idx].ior;
      }
      
      eval_data.k1 = thePrd->wo;
      eval_data.k2 = lightSample.direction;

      optixDirectCall<void>(idxCallScatteringEval, &eval_data, &state, &res_data, nullptr, material.arg_block);

      // This already contains the fabsf(dot(lightSample.direction, state.normal)) factor!
      // For a white Lambert material, the bxdf components match the eval_data.pdf
      const float3 bxdf = eval_data.bsdf_diffuse + eval_data.bsdf_glossy;

      if (0.0f < eval_data.pdf && isNotNull(bxdf))
      {
        // Pass the current payload registers through to the shadow ray.
        unsigned int p0 = optixGetPayload_0();
        unsigned int p1 = optixGetPayload_1();

        thePrd->flags &= ~FLAG_SHADOW; // Clear the shadow flag.

        // Note that the sysData.sceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval 
        // to prevent self-intersections with the actual light geometry in the scene.
        optixTrace(sysData.topObject,
                   thePrd->pos, lightSample.direction, // origin, direction
                   sysData.sceneEpsilon, lightSample.distance - sysData.sceneEpsilon, 0.0f, // tmin, tmax, time
                   OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT, // The shadow ray type only uses anyhit programs.
                   TYPE_RAY_SHADOW, NUM_RAY_TYPES, TYPE_RAY_SHADOW,
                   p0, p1); // Pass through thePrd to the shadow ray.

        if ((thePrd->flags & FLAG_SHADOW) == 0) // Shadow flag not set?
        {
          const float weightMIS = (TYPE_LIGHT_POINT <= light.typeLight) ? 1.0f : balanceHeuristic(lightSample.pdf, eval_data.pdf);
          
          // The sampled emission needs to be scaled by the inverse probability to have selected this light,
          // Selecting one of many lights means the inverse of 1.0f / numLights.
          // This is using the path throughput before the sampling modulated it above.
          thePrd->radiance += throughput * bxdf * lightSample.radiance_over_pdf * (float(numLights) * weightMIS);
        }
      } 
    }
  }

  // Now after everything has been handled using the current material stack,
  // adjust the material stack if there was a transmission crossing a boundary surface.
  if (!thin_walled && (thePrd->eventType & mi::neuraylib::BSDF_EVENT_TRANSMISSION) != 0)
  {
    if (isFrontFace) // Entered a volume. 
    {
      float3 absorption = shaderConfiguration.absorption_coefficient;
      if (0 < shaderConfiguration.idxCallVolumeAbsorptionCoefficient)
      {
        optixDirectCall<void>(shaderConfiguration.idxCallVolumeAbsorptionCoefficient, &absorption, &state, &res_data, nullptr, material.arg_block);
      }

      float3 scattering = shaderConfiguration.scattering_coefficient;
      if (0 < shaderConfiguration.idxCallVolumeScatteringCoefficient)
      {
        optixDirectCall<void>(shaderConfiguration.idxCallVolumeScatteringCoefficient, &scattering, &state, &res_data, nullptr, material.arg_block);
      }

      float bias = shaderConfiguration.directional_bias;
      if (0 < shaderConfiguration.idxCallVolumeDirectionalBias)
      {
        optixDirectCall<void>(shaderConfiguration.idxCallVolumeDirectionalBias, &bias, &state, &res_data, nullptr, material.arg_block);
      }

      const int idx = min(thePrd->idxStack + 1, MATERIAL_STACK_LAST); // Push current medium parameters.

      thePrd->idxStack = idx;
      thePrd->stack[idx].ior     = ior;
      thePrd->stack[idx].sigma_a = absorption;
      thePrd->stack[idx].sigma_s = scattering;
      thePrd->stack[idx].bias    = bias;
      
      thePrd->sigma_t = absorption + scattering; // Update the current extinction coefficient.
    }
    else // if !isFrontFace. Left a volume.
    {
      const int idx = max(0, thePrd->idxStack - 1); // Pop current medium parameters.

      thePrd->idxStack = idx; 

      thePrd->sigma_t = thePrd->stack[idx].sigma_a + thePrd->stack[idx].sigma_s; // Update the current extinction coefficient.
    }
   
    thePrd->walk = 0; // Reset the number of random walk steps taken when crossing any volume boundary.
  }
}


// One anyhit program for the radiance ray for all materials with cutout opacity!
extern "C" __global__ void __anyhit__radiance_cutout()
{
  GeometryInstanceData theData = sysData.geometryInstanceData[optixGetInstanceId()];

  // Cast the hipDeviceptr_t to the actual format for Triangles geometry.
  const unsigned int thePrimitiveIndex = optixGetPrimitiveIndex();

  const uint3* indices = reinterpret_cast<uint3*>(theData.indices);
  const uint3  tri     = indices[thePrimitiveIndex];

  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(theData.attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  const float2 theBarycentrics = optixGetTriangleBarycentrics(); // beta and gamma
  const float  alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;

  float4 objectToWorld[3];
  float4 worldToObject[3];

  getTransforms(optixGetTransformListHandle(0), objectToWorld, worldToObject); // Single instance level transformation list only.

  float3 po = attr0.vertex   * alpha + attr1.vertex   * theBarycentrics.x + attr2.vertex   * theBarycentrics.y;
  float3 ng = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex);
  float3 tg = attr0.tangent  * alpha + attr1.tangent  * theBarycentrics.x + attr2.tangent  * theBarycentrics.y;
  float3 ns = attr0.normal   * alpha + attr1.normal   * theBarycentrics.x + attr2.normal   * theBarycentrics.y;
  
  const float3 tc = attr0.texcoord * alpha + attr1.texcoord * theBarycentrics.x + attr2.texcoord * theBarycentrics.y;

  // Transform into internal space == world space.
  po = transformPoint(objectToWorld,  po);
  ng = normalize(transformNormal(worldToObject, ng));
  tg = normalize(transformVector(objectToWorld, tg));
  ns = normalize(transformNormal(worldToObject, ns));

  TBN tbn(tg, ns); // Calculate an otho-normal system respective to the shading normal.

  Mdl_state state;

  float4 texture_results[16];

  // For explanations of these fields see comments inside __closesthit__radiance above.
  state.normal                = tbn.normal;
  state.geom_normal           = ng;
  state.position              = po;
  state.animation_time        = 0.0f;
  state.text_coords           = &tc;
  state.tangent_u             = &tbn.tangent;
  state.tangent_v             = &tbn.bitangent;
  state.text_results          = texture_results;
  state.ro_data_segment       = nullptr;
  state.world_to_object       = worldToObject;
  state.object_to_world       = objectToWorld;
  state.object_id             = theData.ids.z; // idObject
  state.meters_per_scene_unit = 1.0f;

  const MaterialDefinitionMDL& material = sysData.materialDefinitionsMDL[theData.ids.x];

  mi::neuraylib::Resource_data res_data = { nullptr, material.texture_handler };

  // The cutout opacity value needs to be determined based on the ShaderConfiguration data and geometry.cutout expression when needed.
  const DeviceShaderConfiguration& shaderConfiguration = sysData.shaderConfigurations[material.indexShader];

  // Using a single material init function instead of per distribution init functions.
  // PERF See how that affects cutout opacity which only needs the geometry.cutout expression.
  float opacity = shaderConfiguration.cutout_opacity;

  if (0 <= shaderConfiguration.idxCallGeometryCutoutOpacity)
  {
    // This is always present, even if it just returns.
    optixDirectCall<void>(shaderConfiguration.idxCallInit, &state, &res_data, nullptr, material.arg_block);

    optixDirectCall<void>(shaderConfiguration.idxCallGeometryCutoutOpacity, &opacity, &state, &res_data, nullptr, material.arg_block);
  }

  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  // Stochastic alpha test to get an alpha blend effect.
  // No need to calculate an expensive random number if the test is going to fail anyway.
  if (opacity < 1.0f && opacity <= rng(thePrd->seed))
  {
    optixIgnoreIntersection();
  }
}


// The shadow ray program for all materials with no cutout opacity.
extern "C" __global__ void __anyhit__shadow()
{
  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  // Always set payload values before calling optixIgnoreIntersection or optixTerminateRay because they return immediately!
  thePrd->flags |= FLAG_SHADOW; // Visbility check failed.

  optixTerminateRay();
}


extern "C" __global__ void __anyhit__shadow_cutout() // For the radiance ray type.
{
  GeometryInstanceData theData = sysData.geometryInstanceData[optixGetInstanceId()];

  const unsigned int thePrimitiveIndex = optixGetPrimitiveIndex();

  const uint3* indices = reinterpret_cast<uint3*>(theData.indices);
  const uint3  tri     = indices[thePrimitiveIndex];

  // Cast the hipDeviceptr_t to the actual format for Triangles geometry.
  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(theData.attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  const float2 theBarycentrics = optixGetTriangleBarycentrics(); // beta and gamma
  const float  alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;

  float4 objectToWorld[3];
  float4 worldToObject[3];

  getTransforms(optixGetTransformListHandle(0), objectToWorld, worldToObject); // Single instance level transformation list only.

  float3 po = attr0.vertex   * alpha + attr1.vertex   * theBarycentrics.x + attr2.vertex   * theBarycentrics.y;
  float3 ng = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex);
  float3 tg = attr0.tangent  * alpha + attr1.tangent  * theBarycentrics.x + attr2.tangent  * theBarycentrics.y;
  float3 ns = attr0.normal   * alpha + attr1.normal   * theBarycentrics.x + attr2.normal   * theBarycentrics.y;
  
  const float3 tc = attr0.texcoord * alpha + attr1.texcoord * theBarycentrics.x + attr2.texcoord * theBarycentrics.y;

  // Transform into internal space == world space.
  po = transformPoint(objectToWorld, po);
  ng = normalize(transformNormal(worldToObject, ng));
  tg = normalize(transformVector(objectToWorld, tg));
  ns = normalize(transformNormal(worldToObject, ns));

  TBN tbn(tg, ns); // Calculate an otho-normal system respective to the shading normal.

  Mdl_state state;

  float4 texture_results[16];

  // For explanations of these fields see comments inside __closesthit__radiance above.
  state.normal                = tbn.normal;
  state.geom_normal           = ng;
  state.position              = po;
  state.animation_time        = 0.0f;
  state.text_coords           = &tc;
  state.tangent_u             = &tbn.tangent;
  state.tangent_v             = &tbn.bitangent;
  state.text_results          = texture_results; 
  state.ro_data_segment       = nullptr;
  state.world_to_object       = worldToObject;
  state.object_to_world       = objectToWorld;
  state.object_id             = theData.ids.z; // idObject
  state.meters_per_scene_unit = 1.0f;

  const MaterialDefinitionMDL& material = sysData.materialDefinitionsMDL[theData.ids.x];

  mi::neuraylib::Resource_data res_data = { nullptr, material.texture_handler };

  // The cutout opacity value needs to be determined based on the ShaderConfiguration data and geometry.cutout expression when needed.
  const DeviceShaderConfiguration& shaderConfiguration = sysData.shaderConfigurations[material.indexShader];

  float opacity = shaderConfiguration.cutout_opacity;

  if (0 <= shaderConfiguration.idxCallGeometryCutoutOpacity)
  {
    optixDirectCall<void>(shaderConfiguration.idxCallInit, &state, &res_data, nullptr, material.arg_block);

    optixDirectCall<void>(shaderConfiguration.idxCallGeometryCutoutOpacity, &opacity, &state, &res_data, nullptr, material.arg_block);
  }

  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  // Stochastic alpha test to get an alpha blend effect.
  // No need to calculate an expensive random number if the test is going to fail anyway.
  if (opacity < 1.0f && opacity <= rng(thePrd->seed))
  {
    optixIgnoreIntersection();
  }
  else
  {
    // Always set payload values before calling optixIgnoreIntersection or optixTerminateRay because they return immediately!
    thePrd->flags |= FLAG_SHADOW;

    optixTerminateRay();
  }
}


// Explicit light sampling of a triangle mesh geometry with an emissive MDL material.
// Defined here to be able to use the MDL runtime functions included via texture_lookup.h.
extern "C" __device__ LightSample __direct_callable__light_mesh(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;
 
  lightSample.pdf = 0.0f;

  const float3 sampleTriangle = rng3(prd->seed);

  // Uniformly sample the triangles over their surface area.
  // Note that zero-area triangles (e.g. at the poles of spheres) are automatically never sampled with this method!
  // The cdfU is one bigger than light.width.
  const float* cdfArea = reinterpret_cast<const float*>(light.cdfU);
  const unsigned int idxTriangle = binarySearchCDF(cdfArea, light.width, sampleTriangle.z);

  // Unit square to triangle via barycentric coordinates.
  const float su = sqrtf(sampleTriangle.x);
  // Barycentric coordinates.
  const float alpha = 1.0f - su;
  const float beta  = sampleTriangle.y * su;
  const float gamma = 1.0f - alpha - beta; 
  
  // This cast works because both unsigned int and uint3 have an alignment of 4 bytes.
  const uint3* indices = reinterpret_cast<uint3*>(light.indices);
  const uint3  tri     = indices[idxTriangle];
  
  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(light.attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  float3 po = attr0.vertex   * alpha + attr1.vertex   * beta + attr2.vertex   * gamma;
  float3 ng = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex);
  float3 tg = attr0.tangent  * alpha + attr1.tangent  * beta + attr2.tangent  * gamma;
  float3 ns = attr0.normal   * alpha + attr1.normal   * beta + attr2.normal   * gamma;
  
  const float3 tc = attr0.texcoord * alpha + attr1.texcoord * beta + attr2.texcoord * gamma;

  // Transform into internal space == world space.
  po = transformPoint(light.matrix, po);
  ng = normalize(transformNormal(light.matrixInv, ng));
  tg = normalize(transformVector(light.matrix,    tg));
  ns = normalize(transformNormal(light.matrixInv, ns));

  // Calculate the outgoing direction from light sample position to surface point.
  lightSample.direction = po - prd->pos;  // Sample direction from surface point to light sample position.
  lightSample.distance  = length(lightSample.direction);
    
  if (lightSample.distance < DENOMINATOR_EPSILON)
  {
    return lightSample;
  }

  lightSample.direction *= 1.0f / lightSample.distance; // Normalized vector from light sample position to surface point.
  
  TBN tbn(tg, ns); // Calculate an otho-normal system respective to the shading normal.

  Mdl_state state;

  float4 texture_results[16];

  // For explanations of these fields see comments inside __closesthit__radiance above.
  state.normal                = tbn.normal;
  state.geom_normal           = ng;
  state.position              = po;
  state.animation_time        = 0.0f;
  state.text_coords           = &tc;
  state.tangent_u             = &tbn.tangent;
  state.tangent_v             = &tbn.bitangent;
  state.text_results          = texture_results;
  state.ro_data_segment       = nullptr;
  state.world_to_object       = light.matrixInv;
  state.object_to_world       = light.matrix;
  state.object_id             = light.idObject;
  state.meters_per_scene_unit = 1.0f;

  const MaterialDefinitionMDL& material = sysData.materialDefinitionsMDL[light.idMaterial];

  mi::neuraylib::Resource_data res_data = { nullptr, material.texture_handler };

  const DeviceShaderConfiguration& shaderConfiguration = sysData.shaderConfigurations[material.indexShader];

  // This is always present, even if it just returns.
  optixDirectCall<void>(shaderConfiguration.idxCallInit, &state, &res_data, nullptr, material.arg_block);

  // Arbitrary mesh lights can have cutout opacity!
  float opacity = shaderConfiguration.cutout_opacity;

  if (0 <= shaderConfiguration.idxCallGeometryCutoutOpacity)
  {
    optixDirectCall<void>(shaderConfiguration.idxCallGeometryCutoutOpacity, &opacity, &state, &res_data, nullptr, material.arg_block);
  }

  // If the current light sample is inside a fully cutout region, reject that sample.
  if (opacity <= 0.0f)
  {
    return lightSample;
  }

  // Note that lightSample.direction is from surface point to light sample position.
  const bool isFrontFace = (dot(lightSample.direction, state.geom_normal) < 0.0f);

  // thin_walled value in case the expression was a constant (idxCallThinWalled < 0).
  bool thin_walled = ((shaderConfiguration.flags & IS_THIN_WALLED) != 0);

  if (0 <= shaderConfiguration.idxCallThinWalled)
  {
    optixDirectCall<void>(shaderConfiguration.idxCallThinWalled, &thin_walled, &state, &res_data, nullptr, material.arg_block);
  }

  // Default to no EDF.
  int idxCallEmissionEval          = -1; 
  int idxCallEmissionIntensity     = -1;
  int idxCallEmissionIntensityMode = -1;
  // These are not used when there is no emission, no need to initialize.
  float3 emission_intensity;
  int    emission_intensity_mode;
  
  // MDL Specs: There is no emission on the back-side unless an EDF is specified with the backface field and thin_walled is set to true.
  if (isFrontFace)
  {
    idxCallEmissionEval          = shaderConfiguration.idxCallSurfaceEmissionEval;
    idxCallEmissionIntensity     = shaderConfiguration.idxCallSurfaceEmissionIntensity;
    idxCallEmissionIntensityMode = shaderConfiguration.idxCallSurfaceEmissionIntensityMode;

    emission_intensity      = shaderConfiguration.surface_intensity;
    emission_intensity_mode = shaderConfiguration.surface_intensity_mode;
  }
  else if (thin_walled) // && !isFrontFace
  {
    // These can be the same callable indices if the expressions from surface and backface were identical.
    idxCallEmissionEval          = shaderConfiguration.idxCallBackfaceEmissionEval;
    idxCallEmissionIntensity     = shaderConfiguration.idxCallBackfaceEmissionIntensity;
    idxCallEmissionIntensityMode = shaderConfiguration.idxCallBackfaceEmissionIntensityMode;
    
    emission_intensity      = shaderConfiguration.backface_intensity;
    emission_intensity_mode = shaderConfiguration.backface_intensity_mode;
  }

  // Check if the hit geometry contains any emission.
  if (0 <= idxCallEmissionEval)
  {
    if (0 <= idxCallEmissionIntensity) // Emission intensity is not a constant.
    {
      optixDirectCall<void>(idxCallEmissionIntensity, &emission_intensity, &state, &res_data, nullptr, material.arg_block);
    }
    if (0 <= idxCallEmissionIntensityMode) // Emission intensity mode is not a constant.
    {
      optixDirectCall<void>(idxCallEmissionIntensityMode, &emission_intensity_mode, &state, &res_data, nullptr, material.arg_block);
    }

    if (isNotNull(emission_intensity))
    {
      mi::neuraylib::Edf_evaluate_data<mi::neuraylib::DF_HSM_NONE> eval_data;

      eval_data.k1 = -lightSample.direction; // input: outgoing direction (from light sample position to surface point).
      //eval_data.cos : output: dot(normal, k1)
      //eval_data.edf : output: edf
      //eval_data.pdf : output: pdf (non-projected hemisphere)

      optixDirectCall<void>(idxCallEmissionEval, &eval_data, &state, &res_data, nullptr, material.arg_block);

      // Modulate the emission with the cutout opacity value to get the correct value.
      // The opacity value must not be greater than one here, which could happen for HDR textures.
      opacity = min(opacity, 1.0f);

      // Power (flux) [W] divided by light area gives radiant exitance [W/m^2].
      const float factor = (emission_intensity_mode == 0) ? opacity : opacity / light.area;

      lightSample.pdf = lightSample.distance * lightSample.distance / (light.area * eval_data.cos); // Solid angle measure.

      lightSample.radiance_over_pdf = emission_intensity * eval_data.edf * (factor / lightSample.pdf);
    }
  }

  return lightSample;
}
